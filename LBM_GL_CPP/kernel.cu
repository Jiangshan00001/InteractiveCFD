#include "hip/hip_runtime.h"
#include <string.h>
#include "math.h"
#include "kernel.h"

extern int g_xDim;
extern int g_yDim;
extern int g_paused;

//float uMax = 0.06f;
//float omega = 1.9f;

//int BLOCKSIZEX = 64;
//int BLOCKSIZEY = 1;

//grid and threads for CUDA

//int nBlocks = ((g_xDim + BLOCKSIZEX - 1) / BLOCKSIZEX)*(g_yDim / BLOCKSIZEY);
//int n = nBlocks*BLOCKSIZEX*BLOCKSIZEY;


/*----------------------------------------------------------------------------------------
 *	Device functions
 */

__global__ void UpdateObstructions(Obstruction* obstructions, int obstNumber, float r, float x, float y, Obstruction::Shape shape){
	obstructions[obstNumber].shape = shape;
	obstructions[obstNumber].r1 = r;
	obstructions[obstNumber].x = x;
	obstructions[obstNumber].y = y;
}

inline __device__ bool isInsideObstruction(int x, int y, Obstruction* obstructions, float tolerance = 0.f){
	for (int i = 0; i < MAXOBSTS; i++){
		float r1 = obstructions[i].r1 + tolerance;
		if (obstructions[i].shape == Obstruction::SQUARE){//square
			if (abs(x - obstructions[i].x)<r1 && abs(y - obstructions[i].y)<r1)
				return true;//10;
		}
		else if (obstructions[i].shape == Obstruction::CIRCLE){//circle. shift by 0.5 cells for better looks
			if ((x+0.5f - obstructions[i].x)*(x+0.5f - obstructions[i].x)+(y+0.5f - obstructions[i].y)*(y+0.5f - obstructions[i].y)
					<r1*r1+0.1f)
				return true;//10;
		}
		else if (obstructions[i].shape == Obstruction::HORIZONTAL_LINE){//horizontal line
			if (abs(x - obstructions[i].x)<r1*2 && abs(y - obstructions[i].y)<LINE_OBST_WIDTH*0.501f+tolerance)
				return true;//10;
		}
		else if (obstructions[i].shape == Obstruction::VERTICAL_LINE){//vertical line
			if (abs(y - obstructions[i].y)<r1*2 && abs(x - obstructions[i].x)<LINE_OBST_WIDTH*0.501f+tolerance)
				return true;//10;
		}
	}
	return false;
}

//defines BCs for grid
// no longer in use. 10/29/2016
//inline __device__ int ImageFcn(int x, int y, obstruction* obstructions){
//	//if(y == 0 || x == XDIM-1 || y == YDIM-1)
//	if (x < 0.1f)
//		return 3;//west
//	else if ((XDIM - x) < 1.1f)
//		return 2;//east
//	else if ((YDIM - y) < 1.1f)
//		return 11;//11;//xsymmetry top
//	else if (y < 0.1f)
//		return 12;//12;//xsymmetry bottom

//	for (int i = 0; i < MAXOBSTS; i++){
//		if (abs(x - obstructions[i].x)<obstructions[i].r && abs(y - obstructions[i].y)<obstructions[i].r)
//			return 1;//10;
//	}
//	return 0;
//}

__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b - 1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}
__device__ int dmax(int a, int b)
{
	if (a>b) return a;
	else return b;
}
__device__ float dmin(float a, float b)
{
	if (a<b) return a;
	else return b;
}
__device__ float dmin(float a, float b, float c, float d)
{
	return dmin(dmin(a, b), dmin(c, d));
}
__device__ float dmax(float a)
{
	if (a>0) return a;
	else return 0;
}
__device__ float dmax(float a, float b)
{
	if (a>b) return a;
	else return b;
}
__device__ float dmax(float a, float b, float c, float d)
{
	return dmax(dmax(a, b), dmax(c, d));
}

inline __device__ int f_mem(int f_num, int x, int y, size_t pitch, int yDim)
{

	return (x + y*pitch) + f_num*pitch*yDim;
}

inline __device__ int f_mem(int f_num, int x, int y)
{

	return (x + y*MAX_XDIM) + f_num*MAX_XDIM*MAX_YDIM;
}

__device__ float3 operator+(const float3 &u, const float3 &v)
{
	return make_float3(u.x + v.x, u.y + v.y, u.z + v.z);
}

__device__ float2 operator+(const float2 &u, const float2 &v)
{
	return make_float2(u.x + v.x, u.y + v.y);
}

__device__ float3 operator-(const float3 &u, const float3 &v)
{
	return make_float3(u.x - v.x, u.y - v.y, u.z - v.z);
}

__device__ float2 operator-(const float2 &u, const float2 &v)
{
	return make_float2(u.x - v.x, u.y - v.y);
}

__device__ float3 operator*(const float3 &u, const float3 &v)
{
	return make_float3(u.x * v.x, u.y * v.y, u.z * v.z);
}

__device__ float3 operator/(const float3 &u, const float3 &v)
{
	return make_float3(u.x / v.x, u.y / v.y, u.z / v.z);
}

__device__ float3 operator*(const float a, const float3 &u)
{
	return make_float3(a*u.x, a*u.y, a*u.z);
}

__device__ float DotProduct(float3 u, float3 v)
{
	return u.x*v.x + u.y*v.y + u.z*v.z;
}

__device__ float3 CrossProduct(float3 u, float3 v)
{
	return make_float3(u.y*v.z-u.z*v.y, -(u.x*v.z-u.z*v.x), u.x*v.y-u.y*v.x);
}

__device__ float CrossProductArea(float2 u, float2 v)
{
	return 0.5f*sqrt((u.x*v.y-u.y*v.x)*(u.x*v.y-u.y*v.x));
}

__device__ void Normalize(float3 &u)
{
	float mag = sqrt(DotProduct(u, u));
	u.x /= mag;
	u.y /= mag;
	u.z /= mag;
}

__device__ float Distance(float3 u, float3 v)
{
	float mag = sqrt(DotProduct((u-v), (u-v)));
}

__device__ bool IsPointsOnSameSide(float2 p1, float2 p2, float2 a, float2 b)
{
	float cp1 = (b - a).x*(p1 - a).y - (b - a).y*(p1 - a).x;
	float cp2 = (b - a).x*(p2 - a).y - (b - a).y*(p2 - a).x;
	if (cp1*cp2 >= 0)
	{
		return true;
	}
	return false;
}

__device__ bool IsPointInsideTriangle(float2 p, float2 a, float2 b, float2 c)
{
	if (IsPointsOnSameSide(p, a, b, c) && IsPointsOnSameSide(p, b, a, c) && IsPointsOnSameSide(p, c, a, b))
	{
		return true;
	}
	return false;
}




__device__	void ChangeCoordinatesToNDC(float &xcoord,float &ycoord, int xDimVisible, int yDimVisible)
{
	xcoord = threadIdx.x + blockDim.x*blockIdx.x;
	ycoord = threadIdx.y + blockDim.y*blockIdx.y;
	xcoord /= xDimVisible *0.5f;
	ycoord /= yDimVisible *0.5f;//(float)(blockDim.y*gridDim.y);
	xcoord -= 1.0;// xdim / maxDim;
	ycoord -= 1.0;// ydim / maxDim;
}

__device__	void ChangeCoordinatesToScaledFloat(float &xcoord,float &ycoord, int xDimVisible, int yDimVisible)
{
	xcoord = threadIdx.x + blockDim.x*blockIdx.x;
	ycoord = threadIdx.y + blockDim.y*blockIdx.y;
	xcoord /= xDimVisible *0.5f;
	ycoord /= xDimVisible *0.5f;//(float)(blockDim.y*gridDim.y);
	xcoord -= 1.0;// xdim / maxDim;
	ycoord -= 1.0;// ydim / maxDim;
}

// Initialize domain using constant velocity
__global__ void initialize_single(float4* pos, float *f, int *Im, int xDim, int yDim, float uMax, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	float u, v, rho, usqr;
	rho = 1.f;
	u = uMax;// u_max;// UMAX;
	v = 0.0f;
	usqr = u*u + v*v;

	f[j + 0 * MAX_XDIM*MAX_YDIM] = 0.4444444444f*(rho - 1.5f*usqr);
	f[j + 1 * MAX_XDIM*MAX_YDIM] = 0.1111111111f*(rho + 3.0f*u + 4.5f*u*u - 1.5f*usqr);
	f[j + 2 * MAX_XDIM*MAX_YDIM] = 0.1111111111f*(rho + 3.0f*v + 4.5f*v*v - 1.5f*usqr);
	f[j + 3 * MAX_XDIM*MAX_YDIM] = 0.1111111111f*(rho - 3.0f*u + 4.5f*u*u - 1.5f*usqr);
	f[j + 4 * MAX_XDIM*MAX_YDIM] = 0.1111111111f*(rho - 3.0f*v + 4.5f*v*v - 1.5f*usqr);
	f[j + 5 * MAX_XDIM*MAX_YDIM] = 0.02777777778*(rho + 3.0f*(u + v) + 4.5f*(u + v)*(u + v) - 1.5f*usqr);
	f[j + 6 * MAX_XDIM*MAX_YDIM] = 0.02777777778*(rho + 3.0f*(-u + v) + 4.5f*(-u + v)*(-u + v) - 1.5f*usqr);
	f[j + 7 * MAX_XDIM*MAX_YDIM] = 0.02777777778*(rho + 3.0f*(-u - v) + 4.5f*(-u - v)*(-u - v) - 1.5f*usqr);
	f[j + 8 * MAX_XDIM*MAX_YDIM] = 0.02777777778*(rho + 3.0f*(u - v) + 4.5f*(u - v)*(u - v) - 1.5f*usqr);

	float xcoord, ycoord, zcoord;
	ChangeCoordinatesToScaledFloat(xcoord, ycoord, xDimVisible, yDimVisible);
	zcoord = 0.f;
	float R(255.f), G(255.f), B(255.f), A(255.f);
	char b[] = { R, G, B, A };
	float color;
	std::memcpy(&color, &b, sizeof(color));
	pos[j] = make_float4(xcoord, ycoord, zcoord, color);
}

// rho=1.0 BC for east side
__device__ void NeumannEast(float &f0, float &f1, float &f2,
	float &f3, float &f4, float &f5,
	float &f6, float &f7, float &f8, int y, int xDim, int yDim)
{
	if (y == 0){
		f2 = f4;
		f5 = f8;
	}
	else if (y == yDim - 1){
		f4 = f2;
		f8 = f5;
	}
	float u, v, rho;
	v = 0.0;
	rho = 1.0;
	u = -rho + ((f0 + f2 + f4) + 2.0f*f1 + 2.0f*f5 + 2.0f*f8);

	f3 = f1 - u*0.66666667f;
	f7 = f5 + 0.5f*(f2 - f4) - 0.5f*v - u*0.16666667f;
	f6 = f8 - 0.5f*(f2 - f4) + 0.5f*v - u*0.16666667f;
}

// u=uMax BC for east side
__device__ void DirichletWest(float &f0, float &f1, float &f2,
	float &f3, float &f4, float &f5,
	float &f6, float &f7, float &f8, int y, int xDim, int yDim, float uMax)
{
	if (y == 0){
		f2 = f4;
		f6 = f7;
	}
	else if (y == yDim - 1){
		f4 = f2;
		f7 = f6;
	}
	float u, v;//,rho;
	u = uMax;//*PoisProf(float(y));
	v = 0.0f;//0.0;
	f1 = f3 + u*0.66666667f;
	f5 = f7 - 0.5f*(f2 - f4) + v*0.5f + u*0.166666667f;
	f8 = f6 + 0.5f*(f2 - f4) - v*0.5f + u*0.166666667f;
}

// applies BCs
__device__ void boundaries(float& f0, float& f1, float& f2,
	float& f3, float& f4, float& f5,
	float& f6, float& f7, float& f8,
	int y, int im, int xDim, int yDim, float uMax)
{
	if (im == 2)//NeumannEast
	{
		NeumannEast(f0, f1, f2, f3, f4, f5, f6, f7, f8, y, xDim, yDim);
	}
	else if (im == 3)//DirichletWest
	{
		DirichletWest(f0, f1, f2, f3, f4, f5, f6, f7, f8, y, xDim, yDim, uMax);
	}
	else if (im == 11)//xsymmetry
	{
		f4 = f2;
		f7 = f6;
		f8 = f5;
	}
	else if (im == 12)//xsymmetry
	{
		f2 = f4;
		f6 = f7;
		f5 = f8;
	}
}

// LBM collision step using MRT method
__device__ void mrt_collide(float &f0, float &f1, float &f2,
	float &f3, float &f4, float &f5,
	float &f6, float &f7, float &f8, float omega, float &Q)
{
	//float rho,u,v;	
	float u, v;
	//rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1 - f3 + f5 - f6 - f7 + f8;
	v = f2 - f4 + f5 + f6 - f7 - f8;
	float m1, m2, m4, m6, m7, m8;

	//	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
	m1 = -2.f*f0 + f1 + f2 + f3 + f4 + 4.f*f5 + 4.f*f6 + 4.f*f7 + 4.f*f8 - 3.0f*(u*u + v*v);
	//m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
	m2 = 3.f*f0 - 3.f*f1 - 3.f*f2 - 3.f*f3 - 3.f*f4 + 3.0f*(u*u + v*v); //ep
	//m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
	m4 = -f1 + f3 + 2.f*f5 - 2.f*f6 - 2.f*f7 + 2.f*f8;//-(-u);//qx_eq
	m6 = -f2 + f4 + 2.f*f5 + 2.f*f6 - 2.f*f7 - 2.f*f8;//-(-v);//qy_eq
	m7 = f1 - f2 + f3 - f4 - (u*u - v*v);//pxx_eq
	m8 = f5 - f6 + f7 - f8 - (u*v);//pxy_eq

	//	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
	//	m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
	//	m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
	//	m6 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8-(-v);//qy_eq
	//	m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
	//	m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq
	
	
	float usqr = u*u+v*v;
	float rho = f0 + f1 + f2 + f3 + f4 + f5 + f6 + f7 + f8;
	float feq0 = 4.0f/9.0f*(rho-1.5f*usqr);
	float feq1 = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	float feq2 = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	float feq3 = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	float feq4 = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	float feq5 = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	float feq6 = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	float feq7 = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	float feq8 = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	
	
	float qxx = (f1-feq1) + (f3-feq3) + (f5-feq5) + (f6-feq6) + (f7-feq7) + (f8-feq8);
	float qxy = (f5-feq5) - (f6-feq6) + (f7-feq7) - (f8-feq8)                        ;
	float qyy = (f5-feq5) + (f2-feq2) + (f6-feq6) + (f7-feq7) + (f4-feq4) + (f8-feq8);
	Q = sqrt(qxx*qxx + qxy*qxy * 2 + qyy*qyy);
	float tau0 = 1.f / omega;
	float CS = SMAG_CONST;// 0.1f;
	float tau = 0.5f*tau0 + 0.5f*sqrt(tau0*tau0 + 18.f*CS*sqrt(2.f)*Q);
	omega = 1.f / tau;

	f0 = f0 - (-m1 + m2)*0.11111111f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
	f1 = f1 - (-m1*0.027777777f - 0.05555555556f*m2 - 0.16666666667f*m4 + m7*omega*0.25f);
	f2 = f2 - (-m1*0.027777777f - 0.05555555556f*m2 - 0.16666666667f*m6 - m7*omega*0.25f);
	f3 = f3 - (-m1*0.027777777f - 0.05555555556f*m2 + 0.16666666667f*m4 + m7*omega*0.25f);
	f4 = f4 - (-m1*0.027777777f - 0.05555555556f*m2 + 0.16666666667f*m6 - m7*omega*0.25f);
	f5 = f5 - (0.05555555556f*m1 + m2*0.027777777f + 0.08333333333f*m4 + 0.08333333333f*m6 + m8*omega*0.25f);
	f6 = f6 - (0.05555555556f*m1 + m2*0.027777777f - 0.08333333333f*m4 + 0.08333333333f*m6 - m8*omega*0.25f);
	f7 = f7 - (0.05555555556f*m1 + m2*0.027777777f - 0.08333333333f*m4 - 0.08333333333f*m6 + m8*omega*0.25f);
	f8 = f8 - (0.05555555556f*m1 + m2*0.027777777f + 0.08333333333f*m4 - 0.08333333333f*m6 - m8*omega*0.25f);
}


// main LBM function including streaming and colliding
__global__ void mrt_d_single(float4* pos, float* fA, float* fB,
	float omega, int *Im, Obstruction *obstructions, int contourVar, float contMin, float contMax, int viewMode, int xDim, int yDim, float uMax, int xDimVisible, int yDimVisible)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	int im = Im[j];//ImageFcn(x, y, obstructions); // 
	if (isInsideObstruction(x, y, obstructions)) im = 1;
	float f0, f1, f2, f3, f4, f5, f6, f7, f8;
	f0 = fA[j];
	f1 = fA[f_mem(1, dmax(x - 1), y)];
	f3 = fA[f_mem(3, dmin(x + 1, xDim), y)];
	f2 = fA[f_mem(2, x, y - 1)];
	f5 = fA[f_mem(5, dmax(x - 1), y - 1)];
	f6 = fA[f_mem(6, dmin(x + 1, xDim), y - 1)];
	f4 = fA[f_mem(4, x, y + 1)];
	f7 = fA[f_mem(7, dmin(x + 1, xDim), y + 1)];
	f8 = fA[f_mem(8, dmax(x - 1), dmin(y + 1, yDim))];


	float rho = f0 + f1 + f2 + f3 + f4 + f5 + f6 + f7 + f8;
	float u = f1 - f3 + f5 - f6 - f7 + f8;
	float v = f2 - f4 + f5 + f6 - f7 - f8;
	float usqr = u*u+v*v;
	float StrainRate;

	if (im == 99)
	{
	//do nothing
	}
	else if (im == 1 || im == 10){//bounce-back condition
		//atomicAdd();   //will need this if force is to be computed
		fB[f_mem(1, x, y)] = f3;
		fB[f_mem(2, x, y)] = f4;
		fB[f_mem(3, x, y)] = f1;
		fB[f_mem(4, x, y)] = f2;
		fB[f_mem(5, x, y)] = f7;
		fB[f_mem(6, x, y)] = f8;
		fB[f_mem(7, x, y)] = f5;
		fB[f_mem(8, x, y)] = f6;
	}
	else{
		boundaries(f0, f1, f2, f3, f4, f5, f6, f7, f8, y, im, xDim, yDim, uMax);

		mrt_collide(f0, f1, f2, f3, f4, f5, f6, f7, f8, omega, StrainRate);

		fB[f_mem(0, x, y)] = f0;
		fB[f_mem(1, x, y)] = f1;
		fB[f_mem(2, x, y)] = f2;
		fB[f_mem(3, x, y)] = f3;
		fB[f_mem(4, x, y)] = f4;
		fB[f_mem(5, x, y)] = f5;
		fB[f_mem(6, x, y)] = f6;
		fB[f_mem(7, x, y)] = f7;
		fB[f_mem(8, x, y)] = f8;
	}

	//Prepare data for visualization

	//need to change x,y,z coordinates to NDC (-1 to 1)
	float xcoord, ycoord, zcoord;
	int index;
	//int xdim = blockDim.x*gridDim.x;
	//int ydim = blockDim.y*gridDim.y;
	//xcoord = threadIdx.x + blockDim.x*blockIdx.x;
	//ycoord = threadIdx.y + blockDim.y*blockIdx.y;
	index = j;// x + y*blockDim.x*gridDim.x;
	////	x /= (float)(blockDim.x*gridDim.x)*0.5f;
	////	y /= (float)(blockDim.x*gridDim.x)*0.5f;//(float)(blockDim.y*gridDim.y);
	//xcoord /= xDim / 2;
	//ycoord /= yDim / 2;//(float)(blockDim.y*gridDim.y);
	//xcoord -= 1.0;// xdim / maxDim;
	//ycoord -= 1.0;// ydim / maxDim;

	ChangeCoordinatesToScaledFloat(xcoord, ycoord, xDimVisible, yDimVisible);

	if (im == 1) rho = 1.0;
	zcoord =  (rho - 1.0f) - 0.5f;// *15.f;//f1-f3+f5-f6-f7+f8;//rho;//(rho-1.0f)*2.f;
	//zcoord = -0.5f;// 0.05f*sinf(0.1f*(x)) + 0.05f*sinf(0.1f*y);// (rho - 1.0f) - 0.5f;// *15.f;//f1-f3+f5-f6-f7+f8;//rho;//(rho-1.0f)*2.f;

	//Color c = Color::FromArgb(1);
	//pos[threadIdx.x+threadIdx.y*blockDim.x] = make_float4(x,y,z,1.0f);

	//for color, need to convert 4 bytes (RGBA) to float
	float color;
	float variableValue = 0.f;
	float maxValue;
	float minValue;

	minValue = contMin;
	maxValue = contMax;

	//change min/max contour values based on contour variable
	if (contourVar == ContourVariable::VEL_MAG)
	{
		variableValue = sqrt(u*u+v*v);
	}	
	else if (contourVar == ContourVariable::VEL_U)
	{
		variableValue = u;
	}	
	else if (contourVar == ContourVariable::VEL_V)
	{
		variableValue = v;
	}	
	else if (contourVar == ContourVariable::PRESSURE)
	{
		variableValue = rho;
	}
	else if (contourVar == ContourVariable::STRAIN_RATE)
	{
		variableValue = StrainRate;
	}


	////Blue to white color scheme
	unsigned char R = dmin(255.f,dmax(255 * ((variableValue - minValue) / (maxValue - minValue))));
	unsigned char G = dmin(255.f,dmax(255 * ((variableValue - minValue) / (maxValue - minValue))));
	unsigned char B = 255;// 255 * ((maxValue - variableValue) / (maxValue - minValue));
	unsigned char A = 255;// 255;


	////Rainbow color scheme
	//signed char R = 255 * ((variableValue - minValue) / (maxValue - minValue));
	//signed char G = 255 - 255 * abs(variableValue - 0.5f*(maxValue + minValue)) / (maxValue - 0.5f*(maxValue + minValue));
	//signed char B = 255 * ((maxValue - variableValue) / (maxValue - minValue));
	//signed char A = 255;

	if (contourVar == ContourVariable::WATER_RENDERING)
	{
		variableValue = StrainRate;
		R = 50; G = 120; B = 255;
		A = 155;
	}
//	if (viewMode == ViewMode::THREE_DIMENSIONAL)
//	{
//		A = 155;
//	}

//	if (x >= (xDimVisible))
//	{
//		zcoord = -1.f;
//		R = 0; G = 0; B = 0;
//	}
	if (im == 1){
		R = 204; G = 204; B = 204;
		//zcoord = 0.15f;
	}
	else if (im != 0 || x == xDimVisible-1)
	{
		zcoord = -1.f;
	}
	else
	{
	}


	//if (x > 100 && x < 110 && y > 50 && y < 52)
	//{
	//	R = 255; G = 0; B = 0;
	//}

	
	//char b[] = {(char)R, (char)G, (char)B, (char)A};
	//char b[] = { R*cosTheta, G*cosTheta, B*cosTheta, A };
	char b[] = { R, G, B, A };
	//char b[] = {'100','1','1','100'};
	std::memcpy(&color, &b, sizeof(color));

	//vbo aray to be displayed
	pos[index] = make_float4(xcoord, ycoord, zcoord, color);
	//vel[index] = make_float4(xcoord, ycoord, u, 1.0f);

}

__global__ void CleanUpVBO(float4* pos, int xDimVisible, int yDimVisible)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	if (x >= xDimVisible || y >= yDimVisible)
	{
		unsigned char b[] = { 0,0,0,255 };
		float color;
		std::memcpy(&color, &b, sizeof(color));
		pos[j] = make_float4(pos[j].x, pos[j].y, -1.f, color);
	}
}

__global__ void Lighting(float4* pos, Obstruction *obstructions, int xDimVisible, int yDimVisible, float3 cameraPosition)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	unsigned char color[4];
	std::memcpy(color, &(pos[j].w), sizeof(color));
	float R, G, B, A;
	R = color[0];
	G = color[1];
	B = color[2];
	A = color[3];

	float3 n = { 0, 0, 0 };
	float slope_x = 0.f;
	float slope_y = 0.f;
	float cellSize = 2.f / xDimVisible;
	if (x == 0)
	{
		n.x = -1.f;
	}
	else if (y == 0)
	{
		n.y = -1.f;
	}
	else if (x >= xDimVisible - 1)
	{
		n.x = 1.f;
	}
	else if (y >= yDimVisible - 1)
	{
		n.y = 1.f;
	}
	else if (x > 0 && x < (xDimVisible - 1) && y > 0 && y < (yDimVisible - 1))
	{
		slope_x = (pos[(x + 1) + y*MAX_XDIM].z - pos[(x - 1) + y*MAX_XDIM].z) / (2.f*cellSize);
		slope_y = (pos[(x)+(y + 1)*MAX_XDIM].z - pos[(x)+(y - 1)*MAX_XDIM].z) / (2.f*cellSize);
		n.x = -slope_x*2.f*cellSize*2.f*cellSize;
		n.y = -slope_y*2.f*cellSize*2.f*cellSize;
		n.z = 2.f*cellSize*2.f*cellSize;
	}
	Normalize(n);
	float3 elementPosition = {pos[j].x,pos[j].y,pos[j].z };
	float3 diffuseLightDirection1 = {0.577367, 0.577367, -0.577367 };
	float3 diffuseLightDirection2 = { -0.577367, 0.577367, -0.577367 };
	//float3 cameraPosition = { -1.5, -1.5, 1.5};
	float3 eyeDirection = elementPosition - cameraPosition;
	float3 diffuseLightColor1 = {0.5f, 0.5f, 0.5f};
	float3 diffuseLightColor2 = {0.5f, 0.5f, 0.5f};
	float3 specularLightColor1 = {0.9f, 0.9f, 0.9f};

	float cosTheta1 = -DotProduct(n,diffuseLightDirection1);
	cosTheta1 = cosTheta1 < 0 ? 0 : cosTheta1;
	float cosTheta2 = -DotProduct(n, diffuseLightDirection2);
	cosTheta2 = cosTheta2 < 0 ? 0 : cosTheta2;

	float3 specularLightPosition1 = {-1.5f, -1.5f, 1.5f};
	float3 specularLight1 = elementPosition - specularLightPosition1;
	float3 specularRefection1 = specularLight1 - 2.f*(DotProduct(specularLight1, n)*n);
	Normalize(specularRefection1);
	Normalize(eyeDirection);
	float cosAlpha = -DotProduct(eyeDirection, specularRefection1);
	cosAlpha = cosAlpha < 0 ? 0 : cosAlpha;
	cosAlpha = pow(cosAlpha, 5.f);

	float lightAmbient = 0.3f;
	
	float3 diffuse1  = 0.f*cosTheta1*diffuseLightColor1;
	float3 diffuse2  = 0.f*cosTheta2*diffuseLightColor2;
	float3 specular1 = cosAlpha*specularLightColor1;

	color[0] = color[0]*dmin(1.f,(diffuse1.x+diffuse2.x+specular1.x+lightAmbient));
	color[1] = color[1]*dmin(1.f,(diffuse1.y+diffuse2.y+specular1.y+lightAmbient));
	color[2] = color[2]*dmin(1.f,(diffuse1.z+diffuse2.z+specular1.z+lightAmbient));
	color[3] = A;

	std::memcpy(&(pos[j].w), color, sizeof(color));
}



__global__ void initialize_Floor(float4* pos, float* floor_d, int xDim, int yDim, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = MAX_XDIM*MAX_YDIM + x + y*MAX_XDIM;//index on padded mem (pitch in elements)

	float xcoord, ycoord, zcoord;
	ChangeCoordinatesToScaledFloat(xcoord, ycoord, xDimVisible, yDimVisible);
	zcoord = -1.f;
	unsigned char R(255), G(255), B(255), A(255);

	float3 n = { 0, 0, 1 };
	float slope_x = 0.f;
	float slope_y = 0.f;
	float cellSize = 2.f / xDimVisible;
	if (x == 0)
	{
	}
	else if (y == 0)
	{
	}
	else if (x >= xDimVisible - 1)
	{
	}
	else if (y >= yDimVisible - 1)
	{
	}
	else if (x > 0 && x < (xDimVisible - 1) && y > 0 && y < (yDimVisible - 1))
	{
		slope_x = (pos[(x + 1) + y*MAX_XDIM].z - pos[(x - 1) + y*MAX_XDIM].z) / (2.f*cellSize);
		slope_y = (pos[(x)+(y + 1)*MAX_XDIM].z - pos[(x)+(y - 1)*MAX_XDIM].z) / (2.f*cellSize);
		n.x = -slope_x*2.f*cellSize*2.f*cellSize;
		n.y = -slope_y*2.f*cellSize*2.f*cellSize;
		n.z = 2.f*cellSize*2.f*cellSize;
	}
	Normalize(n);
	float theta1 = acosf(n.z / sqrt(DotProduct(n, n)));
	float theta2 = asinf(1.0 / 1.33f)*sin(theta1);
	float dx = sin(theta1 - theta2)*(pos[(x)+(y)*MAX_XDIM].z + 1.f)*(-n.x);
	float dy = sin(theta1 - theta2)*(pos[(x)+(y)*MAX_XDIM].z + 1.f)*(-n.y);

	float attenuation = 0.1f;// (pos[(x)+(y)*MAX_XDIM].z + 1.f)*0.5f;



	R = 255.f*cos(theta1)*attenuation;
	G = 255.f*cos(theta1)*attenuation;
	B = 255.f*cos(theta1)*attenuation;

	char b[] = { R, G, B, A };
	float color;
	std::memcpy(&color, &b, sizeof(color));
	int newX = x;// +dx;
	int newY = y;// + dy;
	if (newX > 0 && newX < xDimVisible && newY > 0 && newY < yDimVisible && false)
	{
		pos[newX + newY*MAX_XDIM] = make_float4(xcoord, ycoord, zcoord, color);
	}
	else{
		pos[j] = make_float4(xcoord, ycoord, zcoord, color);
	}
}

__global__ void update_Floor(float4* pos, float* floor_d, int xDim, int yDim, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = MAX_XDIM*MAX_YDIM + x + y*MAX_XDIM;//index on padded mem (pitch in elements)

	float xcoord, ycoord, zcoord;
	ChangeCoordinatesToScaledFloat(xcoord, ycoord, xDimVisible, yDimVisible);
	zcoord = -1.f;
	unsigned char R(255), G(255), B(255), A(255);

	float3 n = { 0, 0, 1 };
	float slope_x = 0.f;
	float slope_y = 0.f;
	float cellSize = 2.f / xDimVisible;
	if (x > 0 && x < (xDimVisible - 1) && y > 0 && y < (yDimVisible - 1))
	{
		slope_x = (pos[(x + 1) + y*MAX_XDIM].z - pos[(x - 1) + y*MAX_XDIM].z) / (2.f*cellSize);
		slope_y = (pos[(x)+(y + 1)*MAX_XDIM].z - pos[(x)+(y - 1)*MAX_XDIM].z) / (2.f*cellSize);
		n.x = -slope_x*2.f*cellSize*2.f*cellSize;
		n.y = -slope_y*2.f*cellSize*2.f*cellSize;
		n.z = 2.f*cellSize*2.f*cellSize;
	}
	Normalize(n);
	float theta1 = acosf(n.z / sqrt(DotProduct(n, n)));
	float theta2 = asinf(1.0 / 1.3f*sin(theta1));
	float waterDepth = 80.f;
	float dx = sin(theta1 - theta2)*(pos[(x)+(y)*MAX_XDIM].z + 0.5f)*waterDepth*(-n.x)/sqrt(n.x*n.x + n.y*n.y);
	float dy = sin(theta1 - theta2)*(pos[(x)+(y)*MAX_XDIM].z + 0.5f)*waterDepth*(-n.y)/sqrt(n.x*n.x+n.y*n.y);

	float attenuation = 0.5f;// -0.5f*sin(theta1 - theta2)*(pos[(x)+(y)*MAX_XDIM].z + 0.5f) + 1.f;

	R = 255.f*cos(theta1)*attenuation;
	G = 255.f*cos(theta1)*attenuation;
	B = 255.f*cos(theta1)*attenuation;

	char b[] = { R, G, B, A };
	float color;
	std::memcpy(&color, &b, sizeof(color));
	int newX = x +dx + 0.5f;
	int newY = y + dy+0.5f;
	if (newX > 0 && newX < xDimVisible && newY > 0 && newY < yDimVisible)
	{
		atomicAdd(&floor_d[newX + newY*MAX_XDIM], attenuation);
	}
}

__device__ float2 ComputePositionOfLightOnFloor(float4* pos, float3 incidentLight, int x, int y,  int xDimVisible, int yDimVisible)
{
	int j = MAX_XDIM*MAX_YDIM + x + y*MAX_XDIM;//index on padded mem (pitch in elements)

	unsigned char R(255), G(255), B(255), A(255);

	float3 n = { 0, 0, 1 };
	float slope_x = 0.f;
	float slope_y = 0.f;
	float cellSize = 2.f / xDimVisible;
	if (x > 0 && x < (xDimVisible - 1) && y > 0 && y < (yDimVisible - 1))
	{
		slope_x = (pos[(x + 1) + y*MAX_XDIM].z - pos[(x - 1) + y*MAX_XDIM].z) / (2.f*cellSize);
		slope_y = (pos[(x)+(y + 1)*MAX_XDIM].z - pos[(x)+(y - 1)*MAX_XDIM].z) / (2.f*cellSize);
		n.x = -slope_x*2.f*cellSize*2.f*cellSize;
		n.y = -slope_y*2.f*cellSize*2.f*cellSize;
		n.z = 2.f*cellSize*2.f*cellSize;
	}
	Normalize(n);

	//float2 incidentLightCP, refractedLightCP, normalCP;
	Normalize(incidentLight);
	//incidentLightCP.x = sqrt(incidentLight.x*incidentLight.x + incidentLight.y*incidentLight.y);
	//incidentLightCP.y = incidentLight.z;
	//float incidentLightCP_mag = sqrt(incidentLightCP.x*incidentLightCP.x + incidentLightCP.y*incidentLightCP.y);
	//incidentLightCP.x /= incidentLightCP_mag;
	//incidentLightCP.y /= incidentLightCP_mag;

	//float incidentLightAngleXY = atan2f(incidentLight.y, incidentLight.x);
	//normalCP.x = sqrt(n.x*n.x + n.y*n.y);
	//normalCP.y = n.z;

	//float theta1 = acosf(DotProduct(n,incidentLight));  //incident light is pointing into surface. n is pointing out of surface
	//float theta1 = acosf(n.z / sqrt(DotProduct(n, n)));
	//float theta2 = asinf(1.0 / 1.3f*sin(theta1));
	float waterDepth = 80.f;

	//float rotAngle = theta1 - theta2;
	//if (normalCP.x < incidentLightCP.x) rotAngle = -rotAngle;
	//refractedLightCP.x = incidentLightCP.x*cosf(theta1 - theta2) - incidentLightCP.y*sinf(theta1-theta2);
	//refractedLightCP.y = incidentLightCP.x*cosf(theta1 - theta2) + incidentLightCP.y*sinf(theta1-theta2);

	float3 refractedLight;
	//refractedLight.x = refractedLightCP.x*cosf(incidentLightAngleXY);
	//refractedLight.y = refractedLightCP.x*sinf(incidentLightAngleXY);
	//refractedLight.z = refractedLightCP.y;

	//float gamma = asinf(-incidentLight.z);
	//float alpha = gamma + theta1 - theta2;
	//float deltaFloor;
	//deltaFloor = (pos[(x)+(y)*MAX_XDIM].z + 0.5f)*waterDepth / tanf(alpha);
	//deltaFloor = (pos[(x)+(y)*MAX_XDIM].z + 0.5f)*waterDepth*sinf(PI*0.25f - alpha) / dmax(0.01f, cosf(PI*0.25f - alpha));

	//float dx = deltaFloor*(refractedLight.x) / sqrt(refractedLight.x*refractedLight.x + refractedLight.y*refractedLight.y);
	//float dy = deltaFloor*(refractedLight.y) / sqrt(refractedLight.x*refractedLight.x + refractedLight.y*refractedLight.y);

	float r = 1.0 / 1.3f;
	float c = -(DotProduct(n, incidentLight));
	refractedLight = r*incidentLight + (r*c - sqrt(1.f - r*r*(1.f - c*c)))*n;
	//if (x == 100 && y == 10) printf("light intensity: %ff\n",refractedLight.z);
	//refractedLight = r*(CrossProduct(n,(CrossProduct(-1.f*n,incidentLight))))-1.f*sqrt(1.f-r*r*DotProduct(CrossProduct(n,incidentLight),CrossProduct(n,incidentLight)))*n;


	float dx = -refractedLight.x*(pos[(x)+(y)*MAX_XDIM].z + 1.f)*waterDepth / refractedLight.z;
	float dy = -refractedLight.y*(pos[(x)+(y)*MAX_XDIM].z + 1.f)*waterDepth / refractedLight.z;

	//if (x > 100 && x < 110 && y > 50 && y < 52) printf("%i, %i, %f, %f\n",x,y,dx,dy);

//	float dx = sin(theta1 - theta2)*(pos[(x)+(y)*MAX_XDIM].z + 0.5f)*waterDepth*(-n.x)/sqrt(n.x*n.x + n.y*n.y);
//	float dy = sin(theta1 - theta2)*(pos[(x)+(y)*MAX_XDIM].z + 0.5f)*waterDepth*(-n.y)/sqrt(n.x*n.x+n.y*n.y);

	return float2{ (float)x + dx, (float)y + dy };
}

__device__ float ComputeAreaFrom4Points(float2 nw, float2 ne, float2 sw, float2 se)
{
	float2 vecN = ne - nw;
	float2 vecS = se - sw;
	float2 vecE = ne - se;
	float2 vecW = nw - sw;
	return CrossProductArea(vecN, vecW) + CrossProductArea(vecE, vecS);
}

__global__ void update_LightMesh(float4* pos, float2* lightMesh_d, float3 incidentLight, Obstruction* obstructions, int xDim, int yDim, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	
	if (x < xDimVisible && y < yDimVisible)
	{

	//float3 incidentLight{ 0.f, -0.f, -1.f };
		float2 lightPositionOnFloor;
		if (isInsideObstruction(x, y, obstructions,1.f))
		{
			lightPositionOnFloor = make_float2(x, y);
		}
		else
		{
			lightPositionOnFloor = ComputePositionOfLightOnFloor(pos, incidentLight, x, y, xDimVisible, yDimVisible);
		}

	lightMesh_d[j] = lightPositionOnFloor;
	}
}

__global__ void LightFloorUsingLightMesh(float* floor_d, float2* lightMesh_d, Obstruction* obstructions, int xDim, int yDim, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	if (x < xDimVisible-2 && y < yDimVisible-2)
	{
		float2 nw, ne, sw, se;
		nw = lightMesh_d[(x)+(y+1)*MAX_XDIM];
		ne = lightMesh_d[(x+1)+(y+1)*MAX_XDIM];
		sw = lightMesh_d[(x)+(y)*MAX_XDIM];
		se = lightMesh_d[(x+1)+(y)*MAX_XDIM];
		float areaOfLightMeshOnFloor = ComputeAreaFrom4Points(nw, ne, sw, se);
		float lightIntensity = 0.3f / areaOfLightMeshOnFloor;
		//if (x > 100 && x < 110 && y > 50 && y < 52) lightIntensity = 0.1f;
		//if (areaOfLightMeshOnFloor > 1.f) printf("%f, %f, %f\n",sw.x,sw.y,areaOfLightMeshOnFloor);
		//if (x == 100 && y == 10) printf("light intensity: %ff\n",lightIntensity);

//		nw.x -= 5.f;
//		nw.y += 5.f;
//		sw.x -= 5.f;
//		sw.y -= 5.f;
//		ne.x += 5.f;
//		ne.y += 5.f;
//		se.x += 5.f;
//		se.y -= 5.f;

//		if (x == 100)
//		{
//			lightIntensity = 0.1f;
//		}
//		else
//		{
//			lightIntensity = 0.f;
//		}


//		for (int i = dmax(1.f,dmin(nw.x, ne.x, sw.x, se.x)); i < dmin(xDimVisible-2.f,dmax(nw.x, ne.x, sw.x, se.x)+1.f); i++)
//		{
//			for (int j = dmax(1.f,dmin(nw.y, ne.y, sw.y, se.y)); j < dmin(yDimVisible-2.f,dmax(nw.y, ne.y, sw.y, se.y)+1.f); j++)
//			{
//				if (i >= 0 && i < xDimVisible && j >= 0 && j < yDimVisible)
//				{
//					float2 p = make_float2(i, j);
//					if (IsPointInsideTriangle(p,nw,ne,sw) || IsPointInsideTriangle(p,ne,se,sw))
//					{
//						atomicAdd(&floor_d[i + j*MAX_XDIM], lightIntensity);
//					}
//				}
//			}
//		}
		//floor_d[x + y*MAX_XDIM] = lightIntensity;
		atomicAdd(&floor_d[x + (y)*MAX_XDIM], lightIntensity*0.25f);
		atomicAdd(&floor_d[x+1 + (y)*MAX_XDIM], lightIntensity*0.25f);
		atomicAdd(&floor_d[x+1 + (y+1)*MAX_XDIM], lightIntensity*0.25f);
		atomicAdd(&floor_d[x + (y+1)*MAX_XDIM], lightIntensity*0.25f);
	}
}

__global__ void light_Filter(float* floor_d, float* floorFiltered_d, int xDim, int yDim, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)

	if (x> 1 && y>1 && x < xDimVisible-1 && y<yDimVisible-1)
	{ 
		float light = floor_d[x + y*MAX_XDIM];
		//float light = floor_d[x + y*MAX_XDIM] +floor_d[x + 1 + (y)*MAX_XDIM] + floor_d[x - 1 + (y)*MAX_XDIM] +
		//	floor_d[x + (y + 1)*MAX_XDIM] + floor_d[x + 1 + (y + 1)*MAX_XDIM] + floor_d[x - 1 + (y + 1)*MAX_XDIM] +
		//	floor_d[x + (y - 1)*MAX_XDIM] + floor_d[x + 1 + (y - 1)*MAX_XDIM] + floor_d[x - 1 + (y - 1)*MAX_XDIM];
		floorFiltered_d[j] = light / 9.f;

		//if (x == 100 && y == 10) printf("%f\n",light);
	}
	

}

__global__ void light_Floor(float4* pos, float* floor_d, float* floorFiltered_d, float2* lightMesh_d, Obstruction* obstructions, int xDim, int yDim, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = MAX_XDIM*MAX_YDIM + x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	float xcoord, ycoord, zcoord;

	xcoord = lightMesh_d[x + y*MAX_XDIM].x;
	ycoord = lightMesh_d[x + y*MAX_XDIM].y;
	zcoord = -1.f;

	ChangeCoordinatesToScaledFloat(xcoord, ycoord, xDimVisible, yDimVisible);
	float lightFactor = dmin(1.f,floor_d[x + y*MAX_XDIM]);
	//lightFactor = cosf(x*0.1f);// dmin(1.f, floor_d[x + y*MAX_XDIM]);
	//float lightFactor = floor_d[x + y*MAX_XDIM];
	//if (x == 0 && y == 5) printf("%f\n",floor_d[x + y*MAX_XDIM]);
	floor_d[x + y*MAX_XDIM] = 0.f;

	unsigned char R = 50.f*lightFactor;
	unsigned char G = 120.f*lightFactor;
	unsigned char B = 255.f*lightFactor;
	unsigned char A = 255.f;

	if (isInsideObstruction(x, y, obstructions, 1.f))
	{
		if (isInsideObstruction(x, y, obstructions))
		{
			zcoord = -0.3f;
			lightFactor = 0.8f;
			R = 255.f;
			G = 255.f;
			B = 255.f;
		}
	}

	R *= lightFactor;
	G *= lightFactor;
	B *= lightFactor;

	char b[] = { R, G, B, A };
	float color;
	std::memcpy(&color, &b, sizeof(color));
	pos[j] = make_float4(xcoord, ycoord, zcoord, color);
}
/*----------------------------------------------------------------------------------------
 * End of device functions
 */

__global__ void refraction_Floor(float4* pos, float* floor_d, float* floorFiltered_d, float2* lightMesh_d, Obstruction* obstructions, int xDim, int yDim, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = MAX_XDIM*MAX_YDIM + x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	float xcoord, ycoord, zcoord;

	xcoord = lightMesh_d[x + y*MAX_XDIM].x;
	ycoord = lightMesh_d[x + y*MAX_XDIM].y;

	float2 coordOfFloor = ComputePositionOfLightOnFloor(pos, make_float3(0, 0, -1), x, y, xDimVisible, yDimVisible);
//	float floorX = coordOfFloor.x;
//	float floorY = coordOfFloor.y;

//	float color = floor_d[coordOfFloor.x + coordOfFloor.y*MAX_XDIM];

	ChangeCoordinatesToScaledFloat(xcoord, ycoord, xDimVisible, yDimVisible);
	float lightFactor = cosf(x*0.05f);// dmin(1.f, floor_d[x + y*MAX_XDIM]);
	//float lightFactor = floor_d[x + y*MAX_XDIM];
	//if (x == 0 && y == 5) printf("%f\n",floor_d[x + y*MAX_XDIM]);
	floor_d[x + y*MAX_XDIM] = 0.5f;

	if (isInsideObstruction(x, y, obstructions))
	{
		zcoord = 0.1f;
		lightFactor = 1.f;
	}
	else
	{
		zcoord = -1.f;
	}

	unsigned char R = 50.f*lightFactor;
	unsigned char G = 120.f*lightFactor;
	unsigned char B = 255.f*lightFactor;
	unsigned char A = 255.f;

	char b[] = { R, G, B, A };
	//std::memcpy(&color, &b, sizeof(color));
	//pos[j] = make_float4(xcoord, ycoord, zcoord, color);
}


/*----------------------------------------------------------------------------------------
 * End of device functions
 */
void InitializeDomain(float4* vis, float* f_d, int* im_d, int xDim, int yDim, float uMax, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	//dim3 grid(g_xDim / BLOCKSIZEX, g_yDim / BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	initialize_single << <grid, threads >> >(vis, f_d, im_d, xDim, yDim, uMax, xDimVisible, yDimVisible);
}

void MarchSolution(float4* vis, float* fA_d, float* fB_d, int* im_d, Obstruction* obst_d,
	ContourVariable contVar, float contMin, float contMax, ViewMode viewMode, int xDim, int yDim, float uMax, float omega, int tStep, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	//dim3 grid(g_xDim / BLOCKSIZEX, g_yDim / BLOCKSIZEY);
	for (int i = 0; i < tStep; i++)
	{
		mrt_d_single << <grid, threads >> >(vis, fA_d, fB_d, omega, im_d, obst_d, contVar, contMin, viewMode, contMax, xDim, yDim, uMax, xDimVisible, yDimVisible);
		if (g_paused == 0)
		{
			mrt_d_single << <grid, threads >> >(vis, fB_d, fA_d, omega, im_d, obst_d, contVar, contMin, contMax, viewMode, xDim, yDim, uMax, xDimVisible, yDimVisible);
		}
	}
}

void UpdateDeviceObstructions(Obstruction* obst_d, int targetObstID, Obstruction newObst)
{
	UpdateObstructions << <1, 1 >> >(obst_d,targetObstID,newObst.r1,newObst.x,newObst.y,newObst.shape);
}

void CleanUpDeviceVBO(float4* vis, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(MAX_XDIM / BLOCKSIZEX, MAX_YDIM / BLOCKSIZEY);
	CleanUpVBO << <grid, threads>> >(vis, xDimVisible, yDimVisible);
}

void DeviceLighting(float4* vis, Obstruction* obst_d, int xDimVisible, int yDimVisible, float3 cameraPosition)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	Lighting << <grid, threads>> >(vis, obst_d, xDimVisible, yDimVisible, cameraPosition);
}

void InitializeFloor(float4* vis, float* floor_d, int xDim, int yDim, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	initialize_Floor << <grid, threads >> >(vis, floor_d, xDim, yDim, xDimVisible, yDimVisible);
}

void UpdateFloor(float4* vis, float* floor_d, int xDim, int yDim, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	update_Floor << <grid, threads >> >(vis, floor_d, xDim, yDim, xDimVisible, yDimVisible);
}

void LightFloor(float4* vis, float2* lightMesh_d, float* floor_d, float* floorFiltered_d, Obstruction* obst_d, int xDim, int yDim, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	float3 incidentLight4 = { 0, -20.f, -1.f };
	float3 incidentLight2 = { -0.5f, -0.5f, -1.f };
	float3 incidentLight3 = { -0.5f, 0.5f, -1.f };
	float3 incidentLight1 = { -0.25f, -0.25f, -1.f };
	update_LightMesh << <grid, threads >> >(vis, lightMesh_d, incidentLight1, obst_d, xDim, yDim, xDimVisible, yDimVisible);
	LightFloorUsingLightMesh << <grid, threads >> >(floor_d, lightMesh_d, obst_d, xDim, yDim, xDimVisible, yDimVisible);
	//update_LightMesh << <grid, threads >> >(vis, lightMesh_d, incidentLight2, xDim, yDim, xDimVisible, yDimVisible);
	//LightFloorUsingLightMesh << <grid, threads >> >(floor_d, lightMesh_d, xDim, yDim, xDimVisible, yDimVisible);
	//update_LightMesh << <grid, threads >> >(vis, lightMesh_d, incidentLight3, xDim, yDim, xDimVisible, yDimVisible);
	//LightFloorUsingLightMesh << <grid, threads >> >(floor_d, lightMesh_d, xDim, yDim, xDimVisible, yDimVisible);
	//update_LightMesh << <grid, threads >> >(vis, lightMesh_d, incidentLight4, xDim, yDim, xDimVisible, yDimVisible);
	//LightFloorUsingLightMesh << <grid, threads >> >(floor_d, lightMesh_d, xDim, yDim, xDimVisible, yDimVisible);

	light_Floor << <grid, threads >> >(vis, floor_d, floorFiltered_d, lightMesh_d, obst_d, xDim, yDim, xDimVisible, yDimVisible);


	//light_Filter << <grid, threads >> >(floor_d, floorFiltered_d, xDim, yDim, xDimVisible, yDimVisible);
	//update_LightMesh << <grid, threads >> >(vis, lightMesh_d, incidentLight2, xDim, yDim, xDimVisible, yDimVisible);
	//LightFloorUsingLightMesh << <grid, threads >> >(floor_d, lightMesh_d, xDim, yDim, xDimVisible, yDimVisible);
}

void Refraction(float4* vis, float* floor_d, float* floorFiltered_d, float2* lightMesh_d, int xDim, int yDim, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	//refraction_Floor << <grid, threads >> >(vis, floor_d, floorFiltered_d, lightMesh_d, obst_d, xDim, yDim, xDimVisible, yDimVisible);
}

int runCUDA()
{
    return 0;
}
