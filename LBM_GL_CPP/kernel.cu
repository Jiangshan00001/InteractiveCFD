#include "hip/hip_runtime.h"
#include <string.h>
#include "math.h"
#include "kernel.h"

extern int g_xDim;
extern int g_yDim;

//float uMax = 0.06f;
//float omega = 1.9f;

//int BLOCKSIZEX = 64;
//int BLOCKSIZEY = 1;

//grid and threads for CUDA

//int nBlocks = ((g_xDim + BLOCKSIZEX - 1) / BLOCKSIZEX)*(g_yDim / BLOCKSIZEY);
//int n = nBlocks*BLOCKSIZEX*BLOCKSIZEY;


/*----------------------------------------------------------------------------------------
 *	Device functions
 */

__global__ void UpdateObstructions(Obstruction* obstructions, int obstNumber, float r, float x, float y, Obstruction::Shape shape){
	obstructions[obstNumber].shape = shape;
	obstructions[obstNumber].r1 = r;
	obstructions[obstNumber].x = x;
	obstructions[obstNumber].y = y;
}

inline __device__ bool isInsideObstruction(int x, int y, Obstruction* obstructions){
	for (int i = 0; i < MAXOBSTS; i++){
		if (obstructions[i].shape == Obstruction::SQUARE){//square
			if (abs(x - obstructions[i].x)<obstructions[i].r1 && abs(y - obstructions[i].y)<obstructions[i].r1)
				return true;//10;
		}
		else if (obstructions[i].shape == Obstruction::CIRCLE){//circle. shift by 0.5 cells for better looks
			if ((x+0.5f - obstructions[i].x)*(x+0.5f - obstructions[i].x)+(y+0.5f - obstructions[i].y)*(y+0.5f - obstructions[i].y)
					<obstructions[i].r1*obstructions[i].r1+0.1f)
				return true;//10;
		}
		else if (obstructions[i].shape == Obstruction::HORIZONTAL_LINE){//horizontal line
			if (abs(x - obstructions[i].x)<obstructions[i].r1*2 && abs(y - obstructions[i].y)<LINE_OBST_WIDTH*0.501f)
				return true;//10;
		}
		else if (obstructions[i].shape == Obstruction::VERTICAL_LINE){//vertical line
			if (abs(y - obstructions[i].y)<obstructions[i].r1*2 && abs(x - obstructions[i].x)<LINE_OBST_WIDTH*0.501f)
				return true;//10;
		}
	}
	return false;
}

//defines BCs for grid
// no longer in use. 10/29/2016
//inline __device__ int ImageFcn(int x, int y, obstruction* obstructions){
//	//if(y == 0 || x == XDIM-1 || y == YDIM-1)
//	if (x < 0.1f)
//		return 3;//west
//	else if ((XDIM - x) < 1.1f)
//		return 2;//east
//	else if ((YDIM - y) < 1.1f)
//		return 11;//11;//xsymmetry top
//	else if (y < 0.1f)
//		return 12;//12;//xsymmetry bottom

//	for (int i = 0; i < MAXOBSTS; i++){
//		if (abs(x - obstructions[i].x)<obstructions[i].r && abs(y - obstructions[i].y)<obstructions[i].r)
//			return 1;//10;
//	}
//	return 0;
//}

__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b - 1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}
__device__ float dmin(float a, float b)
{
	if (a<b) return a;
	else return b;
}
__device__ float dmax(float a)
{
	if (a>0) return a;
	else return 0;
}



inline __device__ int f_mem(int f_num, int x, int y, size_t pitch, int yDim)
{

	return (x + y*pitch) + f_num*pitch*yDim;
}

inline __device__ int f_mem(int f_num, int x, int y)
{

	return (x + y*MAX_XDIM) + f_num*MAX_XDIM*MAX_YDIM;
}

__device__ float DotProduct(float3 u, float3 v)
{
	return u.x*v.x + u.y*v.y + u.z*v.z;
}

__device__ float3 CrossProduct(float3 u, float3 v)
{
	return make_float3(u.y*v.z-u.z*v.y, -(u.x*v.z-u.z*v.x), u.x*v.y-u.y*v.x);
}

__device__ void Normalize(float3 &u)
{
	float mag = sqrt(DotProduct(u, u));
	u.x /= mag;
	u.y /= mag;
	u.z /= mag;
}

__device__	void ChangeCoordinatesToNDC(float &xcoord,float &ycoord, int xDimVisible, int yDimVisible)
{
	xcoord = threadIdx.x + blockDim.x*blockIdx.x;
	ycoord = threadIdx.y + blockDim.y*blockIdx.y;
	xcoord /= xDimVisible *0.5f;
	ycoord /= yDimVisible *0.5f;//(float)(blockDim.y*gridDim.y);
	xcoord -= 1.0;// xdim / maxDim;
	ycoord -= 1.0;// ydim / maxDim;
}

__device__	void ChangeCoordinatesToScaledFloat(float &xcoord,float &ycoord, int xDimVisible, int yDimVisible)
{
	xcoord = threadIdx.x + blockDim.x*blockIdx.x;
	ycoord = threadIdx.y + blockDim.y*blockIdx.y;
	xcoord /= xDimVisible *0.5f;
	ycoord /= xDimVisible *0.5f;//(float)(blockDim.y*gridDim.y);
	xcoord -= 1.0;// xdim / maxDim;
	ycoord -= 1.0;// ydim / maxDim;
}

// Initialize domain using constant velocity
__global__ void initialize_single(float4* pos, float *f, int *Im, int xDim, int yDim, float uMax, int xDimVisible, int yDimVisible) //obstruction* obstruction)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	float u, v, rho, usqr;
	rho = 1.f;
	u = uMax;// u_max;// UMAX;
	v = 0.0f;
	usqr = u*u + v*v;

	f[j + 0 * MAX_XDIM*MAX_YDIM] = 0.4444444444f*(rho - 1.5f*usqr);
	f[j + 1 * MAX_XDIM*MAX_YDIM] = 0.1111111111f*(rho + 3.0f*u + 4.5f*u*u - 1.5f*usqr);
	f[j + 2 * MAX_XDIM*MAX_YDIM] = 0.1111111111f*(rho + 3.0f*v + 4.5f*v*v - 1.5f*usqr);
	f[j + 3 * MAX_XDIM*MAX_YDIM] = 0.1111111111f*(rho - 3.0f*u + 4.5f*u*u - 1.5f*usqr);
	f[j + 4 * MAX_XDIM*MAX_YDIM] = 0.1111111111f*(rho - 3.0f*v + 4.5f*v*v - 1.5f*usqr);
	f[j + 5 * MAX_XDIM*MAX_YDIM] = 0.02777777778*(rho + 3.0f*(u + v) + 4.5f*(u + v)*(u + v) - 1.5f*usqr);
	f[j + 6 * MAX_XDIM*MAX_YDIM] = 0.02777777778*(rho + 3.0f*(-u + v) + 4.5f*(-u + v)*(-u + v) - 1.5f*usqr);
	f[j + 7 * MAX_XDIM*MAX_YDIM] = 0.02777777778*(rho + 3.0f*(-u - v) + 4.5f*(-u - v)*(-u - v) - 1.5f*usqr);
	f[j + 8 * MAX_XDIM*MAX_YDIM] = 0.02777777778*(rho + 3.0f*(u - v) + 4.5f*(u - v)*(u - v) - 1.5f*usqr);

	float xcoord, ycoord, zcoord;
	ChangeCoordinatesToScaledFloat(xcoord, ycoord, xDimVisible, yDimVisible);
	zcoord = 0.f;
	float R(255.f), G(255.f), B(255.f), A(255.f);
	char b[] = { R, G, B, A };
	float color;
	std::memcpy(&color, &b, sizeof(color));
	pos[j] = make_float4(xcoord, ycoord, zcoord, color);
}


// rho=1.0 BC for east side
__device__ void NeumannEast(float &f0, float &f1, float &f2,
	float &f3, float &f4, float &f5,
	float &f6, float &f7, float &f8, int y, int xDim, int yDim)
{
	if (y == 0){
		f2 = f4;
		f5 = f8;
	}
	else if (y == yDim - 1){
		f4 = f2;
		f8 = f5;
	}
	float u, v, rho;
	v = 0.0;
	rho = 1.0;
	u = -rho + ((f0 + f2 + f4) + 2.0f*f1 + 2.0f*f5 + 2.0f*f8);

	f3 = f1 - u*0.66666667f;
	f7 = f5 + 0.5f*(f2 - f4) - 0.5f*v - u*0.16666667f;
	f6 = f8 - 0.5f*(f2 - f4) + 0.5f*v - u*0.16666667f;
}

// u=uMax BC for east side
__device__ void DirichletWest(float &f0, float &f1, float &f2,
	float &f3, float &f4, float &f5,
	float &f6, float &f7, float &f8, int y, int xDim, int yDim, float uMax)
{
	if (y == 0){
		f2 = f4;
		f6 = f7;
	}
	else if (y == yDim - 1){
		f4 = f2;
		f7 = f6;
	}
	float u, v;//,rho;
	u = uMax;//*PoisProf(float(y));
	v = 0.0f;//0.0;
	f1 = f3 + u*0.66666667f;
	f5 = f7 - 0.5f*(f2 - f4) + v*0.5f + u*0.166666667f;
	f8 = f6 + 0.5f*(f2 - f4) - v*0.5f + u*0.166666667f;
}

// applies BCs
__device__ void boundaries(float& f0, float& f1, float& f2,
	float& f3, float& f4, float& f5,
	float& f6, float& f7, float& f8,
	int y, int im, int xDim, int yDim, float uMax)
{
	if (im == 2)//NeumannEast
	{
		NeumannEast(f0, f1, f2, f3, f4, f5, f6, f7, f8, y, xDim, yDim);
	}
	else if (im == 3)//DirichletWest
	{
		DirichletWest(f0, f1, f2, f3, f4, f5, f6, f7, f8, y, xDim, yDim, uMax);
	}
	else if (im == 11)//xsymmetry
	{
		f4 = f2;
		f7 = f6;
		f8 = f5;
	}
	else if (im == 12)//xsymmetry
	{
		f2 = f4;
		f6 = f7;
		f5 = f8;
	}
}

// LBM collision step using MRT method
__device__ void mrt_collide(float &f0, float &f1, float &f2,
	float &f3, float &f4, float &f5,
	float &f6, float &f7, float &f8, float omega, float &Q)
{
	//float rho,u,v;	
	float u, v;
	//rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1 - f3 + f5 - f6 - f7 + f8;
	v = f2 - f4 + f5 + f6 - f7 - f8;
	float m1, m2, m4, m6, m7, m8;

	//	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
	m1 = -2.f*f0 + f1 + f2 + f3 + f4 + 4.f*f5 + 4.f*f6 + 4.f*f7 + 4.f*f8 - 3.0f*(u*u + v*v);
	//m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
	m2 = 3.f*f0 - 3.f*f1 - 3.f*f2 - 3.f*f3 - 3.f*f4 + 3.0f*(u*u + v*v); //ep
	//m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
	m4 = -f1 + f3 + 2.f*f5 - 2.f*f6 - 2.f*f7 + 2.f*f8;//-(-u);//qx_eq
	m6 = -f2 + f4 + 2.f*f5 + 2.f*f6 - 2.f*f7 - 2.f*f8;//-(-v);//qy_eq
	m7 = f1 - f2 + f3 - f4 - (u*u - v*v);//pxx_eq
	m8 = f5 - f6 + f7 - f8 - (u*v);//pxy_eq

	//	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
	//	m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
	//	m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
	//	m6 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8-(-v);//qy_eq
	//	m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
	//	m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq
	
	
	float usqr = u*u+v*v;
	float rho = f0 + f1 + f2 + f3 + f4 + f5 + f6 + f7 + f8;
	float feq0 = 4.0f/9.0f*(rho-1.5f*usqr);
	float feq1 = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	float feq2 = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	float feq3 = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	float feq4 = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	float feq5 = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	float feq6 = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	float feq7 = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	float feq8 = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	
	
	float qxx = (f1-feq1) + (f3-feq3) + (f5-feq5) + (f6-feq6) + (f7-feq7) + (f8-feq8);
	float qxy = (f5-feq5) - (f6-feq6) + (f7-feq7) - (f8-feq8)                        ;
	float qyy = (f5-feq5) + (f2-feq2) + (f6-feq6) + (f7-feq7) + (f4-feq4) + (f8-feq8);
	Q = sqrt(qxx*qxx + qxy*qxy * 2 + qyy*qyy);
	float tau0 = 1.f / omega;
	float CS = SMAG_CONST;// 0.1f;
	float tau = 0.5f*tau0 + 0.5f*sqrt(tau0*tau0 + 18.f*CS*sqrt(2.f)*Q);
	omega = 1.f / tau;

	f0 = f0 - (-m1 + m2)*0.11111111f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
	f1 = f1 - (-m1*0.027777777f - 0.05555555556f*m2 - 0.16666666667f*m4 + m7*omega*0.25f);
	f2 = f2 - (-m1*0.027777777f - 0.05555555556f*m2 - 0.16666666667f*m6 - m7*omega*0.25f);
	f3 = f3 - (-m1*0.027777777f - 0.05555555556f*m2 + 0.16666666667f*m4 + m7*omega*0.25f);
	f4 = f4 - (-m1*0.027777777f - 0.05555555556f*m2 + 0.16666666667f*m6 - m7*omega*0.25f);
	f5 = f5 - (0.05555555556f*m1 + m2*0.027777777f + 0.08333333333f*m4 + 0.08333333333f*m6 + m8*omega*0.25f);
	f6 = f6 - (0.05555555556f*m1 + m2*0.027777777f - 0.08333333333f*m4 + 0.08333333333f*m6 - m8*omega*0.25f);
	f7 = f7 - (0.05555555556f*m1 + m2*0.027777777f - 0.08333333333f*m4 - 0.08333333333f*m6 + m8*omega*0.25f);
	f8 = f8 - (0.05555555556f*m1 + m2*0.027777777f + 0.08333333333f*m4 - 0.08333333333f*m6 - m8*omega*0.25f);
}


// main LBM function including streaming and colliding
__global__ void mrt_d_single(float4* pos, float* fA, float* fB,
	float omega, int *Im, Obstruction *obstructions, int contourVar, float contMin, float contMax, int xDim, int yDim, float uMax, int xDimVisible, int yDimVisible)//pitch in elements
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	int im = Im[j];//ImageFcn(x, y, obstructions); // 
	if (isInsideObstruction(x, y, obstructions)) im = 1;
	float f0, f1, f2, f3, f4, f5, f6, f7, f8;
	f0 = fA[j];
	f1 = fA[f_mem(1, dmax(x - 1), y)];
	f3 = fA[f_mem(3, dmin(x + 1, xDim), y)];
	f2 = fA[f_mem(2, x, y - 1)];
	f5 = fA[f_mem(5, dmax(x - 1), y - 1)];
	f6 = fA[f_mem(6, dmin(x + 1, xDim), y - 1)];
	f4 = fA[f_mem(4, x, y + 1)];
	f7 = fA[f_mem(7, dmin(x + 1, xDim), y + 1)];
	f8 = fA[f_mem(8, dmax(x - 1), dmin(y + 1, yDim))];


	float rho = f0 + f1 + f2 + f3 + f4 + f5 + f6 + f7 + f8;
	float u = f1 - f3 + f5 - f6 - f7 + f8;
	float v = f2 - f4 + f5 + f6 - f7 - f8;
	float usqr = u*u+v*v;
	float StrainRate;

	if (im == 99)
	{
	//do nothing
	}
	else if (im == 1 || im == 10){//bounce-back condition
		//atomicAdd();   //will need this if force is to be computed
		fB[f_mem(1, x, y)] = f3;
		fB[f_mem(2, x, y)] = f4;
		fB[f_mem(3, x, y)] = f1;
		fB[f_mem(4, x, y)] = f2;
		fB[f_mem(5, x, y)] = f7;
		fB[f_mem(6, x, y)] = f8;
		fB[f_mem(7, x, y)] = f5;
		fB[f_mem(8, x, y)] = f6;
	}
	else{
		boundaries(f0, f1, f2, f3, f4, f5, f6, f7, f8, y, im, xDim, yDim, uMax);

		mrt_collide(f0, f1, f2, f3, f4, f5, f6, f7, f8, omega, StrainRate);

		fB[f_mem(0, x, y)] = f0;
		fB[f_mem(1, x, y)] = f1;
		fB[f_mem(2, x, y)] = f2;
		fB[f_mem(3, x, y)] = f3;
		fB[f_mem(4, x, y)] = f4;
		fB[f_mem(5, x, y)] = f5;
		fB[f_mem(6, x, y)] = f6;
		fB[f_mem(7, x, y)] = f7;
		fB[f_mem(8, x, y)] = f8;
	}

	//Prepare data for visualization

	//need to change x,y,z coordinates to NDC (-1 to 1)
	float xcoord, ycoord, zcoord;
	int index;
	//int xdim = blockDim.x*gridDim.x;
	//int ydim = blockDim.y*gridDim.y;
	//xcoord = threadIdx.x + blockDim.x*blockIdx.x;
	//ycoord = threadIdx.y + blockDim.y*blockIdx.y;
	index = j;// x + y*blockDim.x*gridDim.x;
	////	x /= (float)(blockDim.x*gridDim.x)*0.5f;
	////	y /= (float)(blockDim.x*gridDim.x)*0.5f;//(float)(blockDim.y*gridDim.y);
	//xcoord /= xDim / 2;
	//ycoord /= yDim / 2;//(float)(blockDim.y*gridDim.y);
	//xcoord -= 1.0;// xdim / maxDim;
	//ycoord -= 1.0;// ydim / maxDim;

	ChangeCoordinatesToScaledFloat(xcoord, ycoord, xDimVisible, yDimVisible);

	if (im == 1) rho = 0.0;
	//zcoord = f1-f3+f5-f6-f7+f8;//rho;//(rho-1.0f)*2.f;
	zcoord = (rho - 1.0f);// *15.f;//f1-f3+f5-f6-f7+f8;//rho;//(rho-1.0f)*2.f;

	//Color c = Color::FromArgb(1);
	//pos[threadIdx.x+threadIdx.y*blockDim.x] = make_float4(x,y,z,1.0f);

	//for color, need to convert 4 bytes (RGBA) to float
	float color;
	float variableValue;
	float maxValue;
	float minValue;

	minValue = contMin;
	maxValue = contMax;

	//change min/max contour values based on contour variable
	if (contourVar == ContourVariable::VEL_MAG)
	{
		variableValue = sqrt(u*u+v*v);
	}	
	else if (contourVar == ContourVariable::VEL_U)
	{
		variableValue = u;
	}	
	else if (contourVar == ContourVariable::VEL_V)
	{
		variableValue = v;
	}	
	else if (contourVar == ContourVariable::PRESSURE)
	{
		variableValue = rho;
	}
	else if (contourVar == ContourVariable::STRAIN_RATE)
	{
		variableValue = StrainRate;
	}

	////Blue to white color scheme
	unsigned char R = dmin(255.f,dmax(255 * ((variableValue - minValue) / (maxValue - minValue))));
	unsigned char G = dmin(255.f,dmax(255 * ((variableValue - minValue) / (maxValue - minValue))));
	unsigned char B = 255;// 255 * ((maxValue - variableValue) / (maxValue - minValue));
	unsigned char A = 255;

	////Rainbow color scheme
	//signed char R = 255 * ((variableValue - minValue) / (maxValue - minValue));
	//signed char G = 255 - 255 * abs(variableValue - 0.5f*(maxValue + minValue)) / (maxValue - 0.5f*(maxValue + minValue));
	//signed char B = 255 * ((maxValue - variableValue) / (maxValue - minValue));
	//signed char A = 255;

	//set walls to be white
	if (x >= xDimVisible)
	{
		zcoord = -1.f;
		R = 0; G = 0; B = 0;
	}
	else if (im == 1){
		R = 204; G = 204; B = 204;
		zcoord = 0.15f;
	}
	//set walls drawn by user to be light gray
	else if (im == 10){
		R = 200; G = 200; B = 200;
	}
	else if (im != 0)
	{
		zcoord = -1.f;
		R = 120; G = 120; B = 255;
	}
	else
	{
		R = 120; G = 120; B = 255;
	}	

	
	//char b[] = {(char)R, (char)G, (char)B, (char)A};
	//char b[] = { R*cosTheta, G*cosTheta, B*cosTheta, A };
	char b[] = { R, G, B, A };
	//char b[] = {'100','1','1','100'};
	std::memcpy(&color, &b, sizeof(color));

	//vbo aray to be displayed
	pos[index] = make_float4(xcoord, ycoord, zcoord, color);
	//vel[index] = make_float4(xcoord, ycoord, u, 1.0f);

}

__global__ void CleanUpVBO(float4* pos, int xDim, int yDim)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	if (x >= xDim || y >= yDim)
	{
		char b[] = { 0,0,0,255 };
		//char b[] = {'100','1','1','100'};
		float color;
		std::memcpy(&color, &b, sizeof(color));
		pos[j] = make_float4(pos[j].x, pos[j].y, -1.f, color);
	}
}

__global__ void Lighting(float4* pos, Obstruction *obstructions, int xDimVisible, int yDimVisible)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int j = x + y*MAX_XDIM;//index on padded mem (pitch in elements)
	unsigned char color[4];
	std::memcpy(color, &(pos[j].w), sizeof(color));
	float R, G, B, A;
	R = color[0];
	G = color[1];
	B = color[2];
	A = color[3];

	float3 n = { 0, 0, 0 };
	float slope_x = 0.f;
	float slope_y = 0.f;
	float cellSize = 2.f / xDimVisible;
	if (x == 0)
	{
		n.x = -1.f;
	}
	else if (y == 0)
	{
		n.y = -1.f;
	}
	else if (x >= xDimVisible - 1)
	{
		n.x = 1.f;
	}
	else if (y >= yDimVisible - 1)
	{
		n.y = 1.f;
	}
	else if (x > 0 && x < (xDimVisible - 1) && y > 0 && y < (yDimVisible - 1))
	{
		slope_x = (pos[(x + 1) + y*MAX_XDIM].z - pos[(x - 1) + y*MAX_XDIM].z) / (2.f*cellSize);
		slope_y = (pos[(x)+(y + 1)*MAX_XDIM].z - pos[(x)+(y - 1)*MAX_XDIM].z) / (2.f*cellSize);
		n.x = -slope_x*2.f*cellSize*2.f*cellSize;
		n.y = -slope_y*2.f*cellSize*2.f*cellSize;
		n.z = 2.f*cellSize*2.f*cellSize;
	}
	Normalize(n);
	float3 l = { 0.577367, 0.577367, -0.577367 };
	float cosTheta = -DotProduct(n,l);
	cosTheta = cosTheta < 0 ? 0 : cosTheta;

	float light_R = 1.f;
	float light_G = 1.f;
	float light_B = 1.f;
	
	color[0] = R*light_R*cosTheta;
	color[1] = G*light_G*cosTheta;
	color[2] = B*light_B*cosTheta;
	color[3] = A;

	std::memcpy(&(pos[j].w), color, sizeof(color));
}


/*----------------------------------------------------------------------------------------
 * End of device functions
 */

void InitializeDomain(float4* vis, float* f_d, int* im_d, int xDim, int yDim, float uMax, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	//dim3 grid(g_xDim / BLOCKSIZEX, g_yDim / BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	initialize_single << <grid, threads >> >(vis, f_d, im_d, xDim, yDim, uMax, xDimVisible, yDimVisible);
}

void MarchSolution(float4* vis, float* fA_d, float* fB_d, int* im_d, Obstruction* obst_d,
	ContourVariable contVar, float contMin, float contMax, int xDim, int yDim, float uMax, float omega, int tStep, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	//dim3 grid(g_xDim / BLOCKSIZEX, g_yDim / BLOCKSIZEY);
	for (int i = 0; i < tStep; i++)
	{
		mrt_d_single << <grid, threads >> >(vis, fA_d, fB_d, omega, im_d, obst_d, contVar, contMin, contMax, xDim, yDim, uMax, xDimVisible, yDimVisible);
		mrt_d_single << <grid, threads >> >(vis, fB_d, fA_d, omega, im_d, obst_d, contVar, contMin, contMax, xDim, yDim, uMax, xDimVisible, yDimVisible);
	}
}

void UpdateDeviceObstructions(Obstruction* obst_d, int targetObstID, Obstruction newObst)
{
	UpdateObstructions << <1, 1 >> >(obst_d,targetObstID,newObst.r1,newObst.x,newObst.y,newObst.shape);
}

void CleanUpDeviceVBO(float4* vis, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(MAX_XDIM / BLOCKSIZEX, MAX_YDIM / BLOCKSIZEY);
	CleanUpVBO << <grid, threads>> >(vis, xDimVisible, yDimVisible);
}

void DeviceLighting(float4* vis, Obstruction* obst_d, int xDimVisible, int yDimVisible)
{
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
	dim3 grid(ceil(static_cast<float>(g_xDim) / BLOCKSIZEX), g_yDim / BLOCKSIZEY);
	Lighting << <grid, threads>> >(vis, obst_d, xDimVisible, yDimVisible);
}

int runCUDA()
{
    return 0;
}
