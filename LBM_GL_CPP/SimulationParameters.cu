#include "hip/hip_runtime.h"
#include "SimulationParameters.h"

SimulationParameters::SimulationParameters()
{
    m_xDim = BLOCKSIZEX * 2;
    m_yDim = BLOCKSIZEX;
    m_xDimVisible = m_xDim;
    m_yDimVisible = m_yDim;
}

__host__ __device__ int SimulationParameters::GetXDim()
{
    return m_xDim;
}

__host__ __device__ int SimulationParameters::GetYDim()
{
    return m_yDim;
}

__host__ __device__ int SimulationParameters::GetXDimVisible()
{
    return m_xDimVisible;
}

__host__ __device__ int SimulationParameters::GetYDimVisible()
{
    return m_yDimVisible;
}

__host__ void SimulationParameters::SetXDim(const int xDim)
{
    //x dimension must be multiple of BLOCKSIZEX
    int xDimAsMultipleOfBlocksize = ceil(static_cast<float>(xDim)/BLOCKSIZEX)*BLOCKSIZEX;
    m_xDim = xDimAsMultipleOfBlocksize < MAX_XDIM ? xDimAsMultipleOfBlocksize : MAX_XDIM;
}

__host__ void SimulationParameters::SetYDim(const int yDim)
{
    //y dimension must be multiple of BLOCKSIZEY
    int yDimAsMultipleOfBlocksize = ceil(static_cast<float>(yDim)/BLOCKSIZEY)*BLOCKSIZEY;
    m_yDim = yDimAsMultipleOfBlocksize < MAX_YDIM ? yDimAsMultipleOfBlocksize : MAX_YDIM;
}

__host__ void SimulationParameters::SetXDimVisible(const int xDimVisible)
{
    m_xDimVisible = xDimVisible < MAX_XDIM ? xDimVisible : MAX_XDIM;
    SetXDim(xDimVisible);
}

__host__ void SimulationParameters::SetYDimVisible(const int yDimVisible)
{
    m_yDimVisible = yDimVisible < MAX_YDIM ? yDimVisible : MAX_YDIM;
    SetYDim(yDimVisible);
}
